
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testFunc()
{
    //int x = blockIdx.x*blockDim.x + threadIdx.x;

    return;
}

__global__ void modifyMeshGPU(float a_fpTime, float* a_pfpMesh)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    if(x<9)
    {
        if(0 == x%3)
        {
            a_pfpMesh[x] = a_pfpMesh[x] + 0.5f * cos(a_fpTime);
        }
    }

    return;
}

extern "C"
void cuda_testFunc()
{
    dim3 dimBlock = dim3(16, 16);
    dim3 dimGrid = dim3(1, 1);

    testFunc<<<dimBlock,dimGrid>>>();

    printf("cuda_testFunc");

    return;
}

extern "C"
void cuda_modifyMesh(float a_fpTime, float* a_pfpMesh)
{
    dim3 dimBlock = dim3(16, 16);
    dim3 dimGrid = dim3(1, 1);

    modifyMeshGPU<<<dimBlock,dimGrid>>>(a_fpTime, a_pfpMesh);
    return;
}